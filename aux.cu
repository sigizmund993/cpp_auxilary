
#include <hip/hip_runtime.h>
#define PI 3.14159265358979323846f
#define ROBOT_R 100.0f
#define OBSTACLE_ANGLE (PI / 5)
#define GOAL_VIEW_ANGLE (PI / 4)
#define DIST_TO_ENEMY 1500
#define SHOOT_ANGLE (PI / 8)
#define DANGER_ZONE_DIST 400
struct Point {
    float x, y;

    __host__ __device__ Point() : x(0), y(0) {}
    __host__ __device__ Point(float x_, float y_) : x(x_), y(y_) {}

    __host__ __device__ Point operator+(Point b) {
        return Point(x + b.x, y + b.y);
    }

    __host__ __device__ Point operator-(Point b) {
        return Point(x - b.x, y - b.y);
    }

    __host__ __device__ Point operator*(float scalar) {
        return Point(x * scalar, y * scalar);
    }
    __host__ __device__ Point operator/(float scalar) {
        return Point(x / scalar, y / scalar);
    }
    __host__ __device__
    bool operator==(Point other) {
        return x == other.x && y == other.y;
    }

    __host__ __device__
    bool operator!=(Point other) {
        return !(*this == other);
    }

    __host__ __device__ float scalar(Point b) {
        return x * b.x + y * b.y;
    }

    __host__ __device__ float vector(Point b) {
        return x * b.y - y * b.x;
    }

    __host__ __device__ float mag() {
        return sqrtf(x * x + y * y);
    }

    __host__ __device__ Point unity() {
        float len = mag();
        return len > 0.0f ? Point(x / len, y / len) : Point(0.0f, 0.0f);
    }

    __host__ __device__ float arg() {
        return atan2f(y, x);
    }
};

struct Field {
    float GOAL_DX, GOAL_DY, ZONE_DX, ZONE_DY, FIELD_DX, FIELD_DY;
    int POLARITY;
    Point hull[4];
    Point enemy_hull[4], ally_hull[4];
    Point enemy_goal[2], ally_goal[2];
    __host__ __device__ Field(float gx, float gy, float zx, float zy, float fy, int pol) {
        GOAL_DX = gx;
        GOAL_DY = gy;
        ZONE_DX = zx;
        ZONE_DY = zy;
        FIELD_DX = gx;
        FIELD_DY = fy;
        POLARITY = pol;
        hull[0] = Point(FIELD_DX, FIELD_DY);
        hull[1] = Point(FIELD_DX, -FIELD_DY);
        hull[2] = Point(-FIELD_DX, -FIELD_DY);
        hull[3] = Point(-FIELD_DX, FIELD_DY);
        enemy_hull[0] = Point(FIELD_DX * POLARITY, ZONE_DY / 2);
        enemy_hull[1] = Point(FIELD_DX * POLARITY, -ZONE_DY / 2);
        enemy_hull[2] = Point((FIELD_DX - ZONE_DX) * POLARITY, -ZONE_DY / 2);
        enemy_hull[3] = Point((FIELD_DX - ZONE_DX) * POLARITY, ZONE_DY / 2);
        ally_hull[0] = Point(FIELD_DX * -POLARITY, ZONE_DY / 2);
        ally_hull[1] = Point(FIELD_DX * -POLARITY, -ZONE_DY / 2);
        ally_hull[2] = Point((FIELD_DX - ZONE_DX) * -POLARITY, -ZONE_DY / 2);
        ally_hull[3] = Point((FIELD_DX - ZONE_DX) * -POLARITY, ZONE_DY / 2);
        enemy_goal[0] = Point(GOAL_DX * POLARITY, GOAL_DY / 2);
        enemy_goal[1] = Point(GOAL_DX * POLARITY, -GOAL_DY / 2);
        ally_goal[0] = Point(GOAL_DX * -POLARITY, GOAL_DY / 2);
        ally_goal[1] = Point(GOAL_DX * -POLARITY, -GOAL_DY / 2);
    }
};

__host__ __device__ int sign(float a) {
    if (a > 0) return 1;
    if (a < 0) return -1;
    return 0;
}

__host__ __device__ float wind_down_angle(float angle) {
    if (fabsf(angle) > 2 * PI) {
        angle = fmodf(angle, 2 * PI);
    }
    if (fabsf(angle) > PI) {
        angle -= 2 * PI * sign(angle);
    }
    return angle;
}

__host__ __device__ float get_angle_between_points(Point a, Point b, Point c) {
    return wind_down_angle((a - b).arg() - (c - b).arg());
}

__host__ __device__ void circles_inter(Point p0, Point p1, float r0, float r1, Point* out) {
    float d = (p0 - p1).mag();
    float a = (r0 * r0 - r1 * r1 + d * d) / (2 * d);
    float h = sqrtf(r0 * r0 - a * a);
    float x2 = p0.x + a * (p1.x - p0.x) / d;
    float y2 = p0.y + a * (p1.y - p0.y) / d;
    out[0].x = x2 + h * (p1.y - p0.y) / d;
    out[0].y = y2 - h * (p1.x - p0.x) / d;
    out[1].x = x2 - h * (p1.y - p0.y) / d;
    out[1].y = y2 + h * (p1.x - p0.x) / d;
}

__host__ __device__ int get_tangent_points(Point point0, Point point1, float r, Point* out) {
    float d = (point1 - point0).mag();
    if (d < r) {
        return 0;
    }   

    if (d == r) {
        out[0] = point1;
        return 1;
    }
    circles_inter(point0, Point((point0.x + point1.x) / 2, (point0.y + point1.y) / 2), r, d / 2, out);
    return 2;
}

__host__ __device__ Point closest_point_on_line(Point point1, Point point2, Point point, char type = 'S') {
    float line_len = (point1 - point2).mag();
    if (line_len == 0) {
        return point1;
    }
    Point line_dir = (point1 - point2).unity();
    Point point_vec = point - point1;
    float dot_product = point_vec.scalar(line_dir);
    if (dot_product <= 0 && type != 'L') {
        return point1;
    }
    if (dot_product >= line_len && type == 'S') {
        return point2;
    }
    return line_dir * dot_product + point1;
}

__host__ __device__ Point nearest_point_on_poly(Point p, Point *poly, int ed_n) {
    float min_ = -1, d;
    Point ans(0, 0), pnt(0, 0);
    for (int i = 0; i < ed_n; i++) {
        pnt = closest_point_on_line(poly[i], poly[i > 0 ? i - 1 : ed_n - 1], p);
        d = (pnt - p).mag();
        if (d < min_ || min_ < 0) {
            min_ = d;
            ans = pnt;
        }
    }
    return ans;
}

__host__ __device__ bool is_point_inside_poly(Point p, Point *points, int ed_n) {
    float old_sign = sign((p - points[ed_n - 1]).vector(points[0] - points[ed_n - 1]));
    for (int i = 0; i < ed_n - 1; i++) {
        if (old_sign != sign((p - points[i]).vector(points[i + 1] - points[i]))) {
            return false;
        }  
    }
    return true;
}

__host__ __device__ Point find_nearest_robot(Point point, Point *team, int te_n) {
    Point ans = Point(0, 0);
    float min_dist = -1, dist;

    if (te_n == 0) {
        return Point(0, 0);
    }
    for (int i = 0; i < te_n; i++) {
        dist = (team[i] - point).mag();
        if (dist < min_dist || min_dist < 0) {
            ans = team[i];
            min_dist = dist;
        }
    }
    return ans;
}

__host__ __device__ float estimate_pass_point(Point *enemies, int en_n, Point frm, Point to) {
    float lerp = 0.0f;
    float ang, ang1, ang2;
    for (int i = 0; i < en_n; i++) {
        float frm_enemy = (enemies[i] - frm).mag();
        if (frm_enemy > ROBOT_R) {
            Point tgs[2];
            get_tangent_points(enemies[i], frm, ROBOT_R, tgs);
            ang1 = get_angle_between_points(to, frm, tgs[0]);
            ang2 = get_angle_between_points(to, frm, tgs[1]);
            ang = fminf(fabsf(ang1), fabsf(ang2));
            if (ang1 * ang2 < 0 && fabsf(ang1) < PI / 2 && fabsf(ang2) < PI / 2)
                    ang *= -1;
        }
        else {
            ang = 2 * asinf(((enemies[i] - to).mag() / 2) / frm_enemy) - asinf(ROBOT_R / frm_enemy);
        }

        if (ang < OBSTACLE_ANGLE) {
            lerp += powf(fabsf((OBSTACLE_ANGLE - ang) / OBSTACLE_ANGLE), 1.5);
        }
            
    }
    return lerp;
}

__host__ __device__ float estimate_goal_view(Point point, Field fld) {
    return fminf(fabsf(get_angle_between_points(fld.enemy_goal[0], point, fld.enemy_goal[1])), 1);
}

__host__ __device__ float estimate_dist_to_boarder(Point point, Field fld) {
    float dist_to_goal_zone = (point - nearest_point_on_poly(point, fld.enemy_hull, 4)).mag();
    if (is_point_inside_poly(point, fld.enemy_goal, 4)) {
        dist_to_goal_zone *= -1;
    }
    
    float dist_to_field_boarder = (point - nearest_point_on_poly(point, fld.hull, 4)).mag();

    float dist_to_danger_zone = fminf(dist_to_goal_zone, dist_to_field_boarder);

    return fmaxf(1 - dist_to_danger_zone / DANGER_ZONE_DIST, 0);
}

__host__ __device__ float estimate_dist_to_enemy(Point point, Point *active_enemies, int en_n) {

    if (en_n == 0) {
        return 0;
    }
    return fmaxf(1 - (find_nearest_robot(point, active_enemies, en_n) - point).mag() / DIST_TO_ENEMY, 0);
}

__host__ __device__ float estimate_shoot(Point point, Field fld, Point *enemies, int en_n) {
    float lerp = 0.0f;
    float ang, ang1, ang2;
    float frm_enemy;
    for (int i = 0; i < en_n; i++) {
        frm_enemy = (point - enemies[i]).mag();
        if (frm_enemy > ROBOT_R) {
            ang1 = get_angle_between_points(fld.enemy_goal[0], point, enemies[i]);
            ang2 = get_angle_between_points(fld.enemy_goal[1], point, enemies[i]);

            ang = fminf(fabsf(ang1), fabsf(ang2));

            if (ang < SHOOT_ANGLE) {
                lerp += powf(fabsf((SHOOT_ANGLE - ang) / SHOOT_ANGLE), 1.5);
            }
        }
    }
    return lerp;
}

__host__ __device__ float estimate_point(Field fld, Point point, Point kick_point, Point *enemies, int en_n) {
    return estimate_goal_view(point, fld) - estimate_pass_point(enemies, en_n, kick_point, point) - estimate_dist_to_boarder(point, fld) - 
    estimate_shoot(point, fld, enemies, en_n) - estimate_dist_to_enemy(point, enemies, en_n);

}